#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "Bot.h"
#include "Individual.h"
#include "NeuralNetwork.h"
#include "Generation.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include ""
#include <hip/device_functions.h>
#include "Battle.h"
#include "Arena.h"
#include <math.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#define SMCount 13
#define NetworksPerBlock 10

__global__ void OneOnOneBattles(float* pw, float* pt, DNeuralNetwork* nnmodel, int* weightCount, int* networkCount, int* battleCount, float* resultsPersonal, float* resultsLegacy, int* nnIDs) {
	__shared__ DNeuralNetwork nnModel = *(nnmodel);
	__syncthreads();
	__shared__ DNeuralNetwork* nns;
	int plus = 0;
	if (blockIdx.x >= (*(networkCount) % SMCount)) plus = 1;
	if (threadIdx.x + plus <= (*(networkCount) / SMCount + 1) || (threadIdx.x < NetworksPerBlock && threadIdx.x < *networkCount)) {
		nns[threadIdx.x] = NNextractFromWeightArray(pw, pt, (blockIdx.x + SMCount * threadIdx.x) % *networkCount, nnModel, getID((blockIdx.x + SMCount * threadIdx.x) % *networkCount, nnIDs));
	}
	int nnCount = (int)(*(networkCount) / (SMCount + 1)) + 1 - plus;
	printf("test");
	__syncthreads();
	__shared__ int responsibleNetworks = nnCount;
	__syncthreads();
	if (responsibleNetworks == 0) return;
	
	__syncthreads();

	DNeuralNetwork contender;
	DNeuralNetwork opponent;
	int hasFight = 0;
	if (threadIdx.x < responsibleNetworks * *battleCount) {
		hasFight = 1;
		contender = nns[threadIdx.x / *battleCount];
		opponent = nns[threadIdx.x / *battleCount + threadIdx.x % (nnCount - 1)];


		float* pf1 = getPersonalFitness(resultsPersonal, contender.id, *networkCount, nnIDs);
		float* lf1 = getPersonalFitness(resultsLegacy, contender.id, *networkCount, nnIDs);
		float* pf2 = getPersonalFitness(resultsPersonal, opponent.id, *networkCount, nnIDs);
		float* lf2 = getPersonalFitness(resultsLegacy, opponent.id, *networkCount, nnIDs);
		
		int* result = battle(contender, opponent);
		if (result[1] == -2) {
			double dFitness = abs(*pf1 - *pf2);

			if (result[1] == 1) {
				//System.out.print("   CONTESTANT won.");
				double skillBoost = 0;
				if (*pf1 > *pf2) {
					skillBoost = 1 / (1 + dFitness / 3);
				}
				else {
					skillBoost = dFitness / 5;
				}
				*pf1 += (skillBoost + 1) * (result[0] / START_HEALTH * 5);
				*pf2 -= (skillBoost + 1) * (result[0] / START_HEALTH * 5);
			}
			else { //(botb.getID() == winner)
				if (result[1] == -2) {
					//System.out.print("   NOONE    won.");
				}
				else {
					//System.out.print("   " + g.ids[opponent].name + " won.");	
				}
				double skillBoost = 0;
				if (*pf2 > *pf1) {
					skillBoost = 1 / (1 + dFitness / 3);
				}
				else {
					skillBoost = dFitness / 5;
				}
				*pf2 += ((skillBoost + 1) * (result[0] / START_HEALTH * 5));
				*pf1 += ((skillBoost + 1) * (result[0] / START_HEALTH * 5 - 1));
			}

			//Update legacy fitness
			dFitness = dFitness = abs(*lf1 - *lf2);
			if (result[1] == 1) {
				double skillBoost = 0;
				if (*lf1 > *lf2) {
					skillBoost = 1 / (1 + dFitness / 3);
				}
				else {
					skillBoost = dFitness / 5;
				}
				*lf1 += (skillBoost + 1) * (result[0] / START_HEALTH * 5);
				*lf2 -= (skillBoost + 1) * (result[0] / START_HEALTH * 5);
			}
			else { //(botb.getID() == winner)
				double skillBoost = 0;
				if (*lf2 > *lf1) {
					skillBoost = 1 / (1 + dFitness / 3);
				}
				else {
					skillBoost = dFitness / 5;
				}
				*lf2 += ((skillBoost + 1) * (result[0] / START_HEALTH * 5));
				*lf1 += ((skillBoost + 1) * (result[0] / START_HEALTH * 5 - 1));
			}
		}
	}
}

__device__ int* battle(DNeuralNetwork nncontender, DNeuralNetwork nnopponent) {
	DNeuralBot b1 = DNeuralBot(&nncontender);
	DNeuralBot b2 = DNeuralBot(&nnopponent);
	DArena a = DArena(b1, b2);
	int result[2];

	int timeout = 100000;
	int count = 0;
	while (count < timeout) {
		AUpdate(a);
		if (a.winner == -2) {
			result[0] = abs(b1.health - b2.health);
			result[1] = -2;
			return  result;
		}
		if (a.winner == b1.id) {
			result[0] = abs(b1.health - b2.health);
			result[1] = 1;
			return  result;
		}
		if (a.winner == b2.id) {
			result[0] = abs(b1.health - b2.health);
			result[1] = 2;
			return  result;
		}

		count++;
	}
	result[0] = -1;
	result[1] = -1;
	return result;
}

__device__ float* getPersonalFitness(float* pf, int ID, int count, int* ids) {
	for (int i = 0; i < count; i++) {
		if (*(ids + i) == ID) return pf + i;
	}
	//Error
	return pf;
}
__device__ float* getLegacyFitness(float* lf, int ID, int count, int* ids) {
	for (int i = 0; i < count; i++) {
		if (*(ids + i) == ID) return lf + i;
	}
	//Error
	return lf;
}

void SetUpOneOnOneBattle(HGeneration g, int battlesPerIndividual) {

	thrust::host_vector<float> hpersonalFitness;
	thrust::host_vector<float> hlegacyFitness;
	thrust::host_vector<float> hnnIDs;
	thrust::host_vector<float> hweights;
	thrust::host_vector<float> hthresh;

	for (int i = 0; i < g.ids->size; i++) {
		hpersonalFitness.push_back((*g.ids)[i].personalFitness);
		hlegacyFitness.push_back((*g.ids)[i].legacyFitness);
		hnnIDs.push_back((*g.ids)[i].nnwrk.id);

		for (int j = 0; j < (*g.ids)[i].nnwrk.layers->size; i++) {
			for (int k = 0; k < (*(*g.ids)[i].nnwrk.layers)[j].neurons->size; k++) {
				hthresh.push_back((*(*(*g.ids)[i].nnwrk.layers)[j].neurons)[k].thresh);
				for (int l = 0; l < (*(*(*g.ids)[i].nnwrk.layers)[j].neurons)[k].w->size; l++) {
					hweights.push_back((*(*(*(*g.ids)[i].nnwrk.layers)[j].neurons)[k].w)[l]);
				}
			}
		}
	}


	float* h_w = thrust::raw_pointer_cast(&hweights[0]);
	float* h_t = thrust::raw_pointer_cast(&hthresh[0]);
	float* h_rp = thrust::raw_pointer_cast(&hpersonalFitness[0]);
	float* h_rl = thrust::raw_pointer_cast(&hlegacyFitness[0]);
	float* h_nnids = thrust::raw_pointer_cast(&hnnIDs[0]);;
	int weightCount = hweights.size();
	
	float* d_w;
	float* d_t;
	float* d_rp;
	float* d_rl;
	float* d_nnids;
	int * d_weightCount;
	int * d_networkCount;
	int * d_battleCount;
	DNeuralNetwork *d_nnmodel;


	for (int i = 0; i < hpersonalFitness.size; i++) {
		std::cout << i << " Personal fitness before: " << hpersonalFitness[i] << std::endl;
	}

	hipMalloc(&d_w, sizeof(float) * hweights.size());
	hipMalloc(&d_t, sizeof(float) * hthresh.size());
	hipMalloc(&d_rp, sizeof(float) * hpersonalFitness.size());
	hipMalloc(&d_rl, sizeof(float) * hlegacyFitness.size());
	hipMalloc(&d_nnids, sizeof(int) * hnnIDs.size());
	hipMalloc(&d_nnmodel, sizeof(HNeuralNetwork));
	hipMalloc(&d_weightCount, sizeof(int));
	hipMalloc(&d_networkCount, sizeof(int));
	hipMalloc(&d_battleCount, sizeof(int));

	hipMemcpy(d_w, h_w, sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_t, h_t, sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_nnmodel, &((*g.ids)[0].nnwrk), sizeof(HNeuralNetwork), hipMemcpyHostToDevice);
	hipMemcpy(d_weightCount, &weightCount, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_networkCount, &g.ids->size, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_battleCount, &battlesPerIndividual, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_rp, h_rp, sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_rl, h_rl, sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_nnids, h_nnids, sizeof(int*), hipMemcpyHostToDevice);

	OneOnOneBattles << <13, 128 >> (d_pw, d_pt, d_nnmodel, d_wegithCount, d_networkCount, d_battleCount, d_rp, d_rl, d_nnids);

	hipMemcpy(h_rp, d_rp, sizeof(float) * hpersonalFitness.size(), hipMemcpyDeviceToHost);
	hipMemcpy(h_rl, d_rl, sizeof(float) * hlegacyFitness.size(), hipMemcpyDeviceToHost);

	hipFree(d_w);
	hipFree(d_t);
	hipFree(d_nnmodel);
	hipFree(d_weightCount);
	hipFree(d_networkCount);
	hipFree(d_battleCount);
	hipFree(d_rp);
	hipFree(d_rl);
	hipFree(d_nnids);

	for (int i = 0; i < hpersonalFitness.size; i++) {
		std::cout << i << " Personal fitness after: " << hpersonalFitness[i] << std::endl;
	}	
}